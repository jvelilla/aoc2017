
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>
#include <iomanip>
#include <iostream>

int const MARKS = 256;
int const ROWS = 128;

__global__
void knotHash(unsigned char const *input, int inputSize, int *usedSquares) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned char lengths[64];
  int numLengths = 0;
  for (int i = 0; i < inputSize; i++) {
    lengths[numLengths++] = input[i];
  }
  lengths[numLengths++] = '-';
  if (row >= 100) {
    lengths[numLengths++] = '0' + (row / 100);
  }
  if (row >= 10) {
    lengths[numLengths++] = '0' + ((row % 100) / 10);
  }
  lengths[numLengths++] = '0' + (row % 10);
  unsigned char const APPEND[] = { 17, 31, 73, 47, 23 };
  int const APPEND_SIZE = sizeof(APPEND) / sizeof(unsigned char);
  for (int i = 0; i < APPEND_SIZE; i++) {
    lengths[numLengths++] = APPEND[i];
  }

  int items[MARKS];
  for (int i = 0; i < MARKS; i++) {
    items[i] = i;
  }

  int start = 0;
  int skip = 0;
  for (int round = 0; round < 64; round++) {
    for (int i = 0; i < numLengths; i++) {
      int length = lengths[i];
      for (int j = 0; j < length / 2; j++) {
        int a = (start + j) % MARKS;
        int b = (start + length - 1 - j) % MARKS;
        unsigned char t = items[a];
        items[a] = items[b];
        items[b] = t;
      }
      start = (start + length + skip) % 256;
      skip = (skip + 1) % 256;
    }
  }

  unsigned char hash[16];
  for (int i = 0; i < 16; i++) {
    unsigned char xored = 0;
    for (int j = 0; j < 16; j++) {
      xored ^= items[16 * i + j];
    }
    hash[i] = xored;
  }

  int bitCount = 0;
  for (int i = 0; i < 16; i++) {
    for (int j = 1; j < 0x100; j <<= 1) {
      if (hash[i] & j) {
        bitCount++;
      }
    }
  }
  usedSquares[row] = bitCount;
}

int main(void) {
  std::string inputStr;
  std::getline(std::cin, inputStr);
  int n = inputStr.size();

  unsigned char *input;
  hipMallocManaged(&input, n);
  memcpy(input, inputStr.data(), n);

  int *usedSquares;
  hipMallocManaged(&usedSquares, ROWS * sizeof(int));

  knotHash<<<1, ROWS>>>(input, n, usedSquares);

  hipDeviceSynchronize();

  int totalUsedSquares = 0;
  for (int i = 0; i < ROWS; i++) {
    totalUsedSquares += usedSquares[i];
  }
  std::cout << totalUsedSquares << '\n';

  hipFree(input);
  hipFree(usedSquares);

  return 0;
}
