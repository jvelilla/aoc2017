
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstring>
#include <iomanip>
#include <iostream>

int const MARKS = 256;
int const ROWS = 128;
int const COLS = 128;

__global__
void knotHash(unsigned char const *input, int inputSize, int *grid) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned char lengths[64];
  int numLengths = 0;
  for (int i = 0; i < inputSize; i++) {
    lengths[numLengths++] = input[i];
  }
  lengths[numLengths++] = '-';
  if (row >= 100) {
    lengths[numLengths++] = '0' + (row / 100);
  }
  if (row >= 10) {
    lengths[numLengths++] = '0' + ((row % 100) / 10);
  }
  lengths[numLengths++] = '0' + (row % 10);
  unsigned char const APPEND[] = { 17, 31, 73, 47, 23 };
  int const APPEND_SIZE = sizeof(APPEND) / sizeof(unsigned char);
  for (int i = 0; i < APPEND_SIZE; i++) {
    lengths[numLengths++] = APPEND[i];
  }

  int items[MARKS];
  for (int i = 0; i < MARKS; i++) {
    items[i] = i;
  }

  int start = 0;
  int skip = 0;
  for (int round = 0; round < 64; round++) {
    for (int i = 0; i < numLengths; i++) {
      int length = lengths[i];
      for (int j = 0; j < length / 2; j++) {
        int a = (start + j) % MARKS;
        int b = (start + length - 1 - j) % MARKS;
        unsigned char t = items[a];
        items[a] = items[b];
        items[b] = t;
      }
      start = (start + length + skip) % 256;
      skip = (skip + 1) % 256;
    }
  }

  unsigned char hash[16];
  for (int i = 0; i < 16; i++) {
    unsigned char xored = 0;
    for (int j = 0; j < 16; j++) {
      xored ^= items[16 * i + j];
    }
    hash[i] = xored;
  }

  for (int i = 0; i < COLS; i++) {
    int index = COLS * row + i;
    grid[index] = (hash[i / 8] & (1 << (7 - (i % 8)))) ? -1 : 0;
  }
}

class Fifo {
  static int const SIZE = ROWS * COLS;
  int elements[SIZE];
  int head;
  int tail;

public:
  __device__
  Fifo() {
    head = 0;
    tail = 0;
  }

  __device__
  void add(int element) {
    elements[tail] = element;
    tail = (tail + 1) % SIZE;
  }

  __device__
  int remove() {
    int element = elements[head];
    head = (head + 1) % SIZE;
    return element;
  }

  __device__
  bool isEmpty() const {
    return head == tail;
  }
};

__device__
int idx(int i, int j) {
  return i * COLS + j;
}

__device__
void floodFill(int *grid, int i, int j, int label) {
  Fifo queue;
  queue.add(idx(i, j));
  while (!queue.isEmpty()) {
    int index = queue.remove();
    if (grid[index] >= 0) {
      continue;
    }
    grid[index] = label;
    int i = index / COLS;
    int j = index % COLS;
    if (i > 0) queue.add(idx(i - 1, j));
    if (i < ROWS - 1) queue.add(idx(i + 1, j));
    if (j > 0) queue.add(idx(i, j - 1));
    if (j < COLS - 1) queue.add(idx(i, j + 1));
  }
}

__global__
void labelConnectedComponents(int *grid, int *numConnectedComponents) {
  int numLabels = 0;
  for (int i = 0; i < ROWS; i++) {
    for (int j = 0; j < COLS; j++) {
      if (grid[i * COLS + j] < 0) {
        numLabels++;
        floodFill(grid, i, j, numLabels);
      }
    }
  }
  *numConnectedComponents = numLabels;
}

int main(void) {
  std::string inputStr;
  std::getline(std::cin, inputStr);
  int n = inputStr.size();

  unsigned char *input;
  hipMallocManaged(&input, n);
  memcpy(input, inputStr.data(), n);

  int *grid;
  hipMallocManaged(&grid, ROWS * COLS * sizeof(int));

  int *numConnectedComponents;
  hipMallocManaged(&numConnectedComponents, sizeof(int));

  hipDeviceSetLimit(hipLimitStackSize, 10 * (1 << 20));

  knotHash<<<1, ROWS>>>(input, n, grid);
  hipDeviceSynchronize(); // Not sure if needed to prevent both kernels running in parallel.
  labelConnectedComponents<<<1, 1>>>(grid, numConnectedComponents);
  hipDeviceSynchronize();

  std::cout << *numConnectedComponents << '\n';

  hipFree(input);
  hipFree(grid);
  hipFree(numConnectedComponents);

  return 0;
}

